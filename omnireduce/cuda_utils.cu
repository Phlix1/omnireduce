#include "hip/hip_runtime.h"
#include "cuda_utils.hpp"

#define CUDA_CALL(x) do { \
      hipError_t _m_cudaStat = x; \
      if((_m_cudaStat) != hipSuccess) { \
      fprintf(stderr, "Error %s at line %d in file %s", \
      hipGetErrorString(_m_cudaStat), __LINE__, __FILE__); \
      exit(1);}} while(0)

#define CURAND_CALL(x) do { if((x) != HIPRAND_STATUS_SUCCESS) { \
      printf("CURAND rrror at %s:%d\n",__FILE__,__LINE__);            \
      exit(1);}} while(0)

template <typename scalar_t>
__global__ void bitmap_cuda_kernel(scalar_t* input, uint8_t* bitmap, int64_t len, scalar_t threshold) {
    const auto index = blockIdx.x * blockDim.x + threadIdx.x;
    __shared__ bool zero_block;
    if (threadIdx.x == 0) zero_block = true;
    __syncthreads();
    if(index < len) {
      if(std::abs(input[index]) > threshold) zero_block=false;
    }
    __syncthreads();
    if(index < len) {
      if(zero_block) {
        input[index]=(scalar_t)0;
        bitmap[blockIdx.x]=1;
      }
      else {
        bitmap[blockIdx.x]=0;
      }
    }
    __syncthreads();
}

void compute_bitmap(float* d_tensor, uint8_t* d_bitmap, int64_t tensor_size, uint32_t block_size, hipStream_t stream, float threshold) {
    uint32_t block_num = tensor_size/block_size;
    if (tensor_size%block_size!=0)
        block_num += 1;
    bitmap_cuda_kernel<<<block_num, block_size, 0, stream>>>(d_tensor, d_bitmap, tensor_size, threshold);
}

void compute_bitmap(int* d_tensor, uint8_t* d_bitmap, int64_t tensor_size, uint32_t block_size, hipStream_t stream, int threshold) {
  uint32_t block_num = tensor_size/block_size;
  if (tensor_size%block_size!=0)
      block_num += 1;
  bitmap_cuda_kernel<<<block_num, block_size, 0, stream>>>(d_tensor, d_bitmap, tensor_size, threshold);
}

void compute_bitmap(uint8_t* d_tensor, uint8_t* d_bitmap, int64_t tensor_size, uint32_t block_size, hipStream_t stream, uint8_t threshold) {
  uint32_t block_num = tensor_size/block_size;
  if (tensor_size%block_size!=0)
      block_num += 1;
  bitmap_cuda_kernel<<<block_num, block_size, 0, stream>>>(d_tensor, d_bitmap, tensor_size, threshold);
}

__global__ void cnat_compress_cuda_kernel(
    float* __restrict__ input,
    uint8_t* __restrict__ output,
    int len) {
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  if(index < len) {
    if (input[index] == 0)
      output[index] = 0;
    else {
      int rand_bits = output[index];
      rand_bits <<= 15;
      int exp;
      int prob = reinterpret_cast<int &>(input[index]) & 0b00000000011111111000000000000000;
      frexpf(input[index], &exp);
      if (rand_bits >= prob) exp -= 1;
      exp += 127;
      uint8_t encode;
      if (exp<=17) encode = 0;
      else if (exp<=143) encode = uint8_t(exp-17);
      else encode = 127;
      if (input[index] < 0) encode += 128;
      output[index] = encode;
    }
  }
}

/*
  generate random bits using the output buffer (8 bits for each element)
  and compare with the 8 most significant bits of mantissa
*/
void cnat_compress(float* input, uint8_t* output, int count, hipStream_t stream, hiprandGenerator_t* gen) {
    const int threads = 1024;
    auto blocks = count/threads;
    if (count%threads || !blocks) blocks++;
    CURAND_CALL(hiprandSetStream(*gen, stream));
    CURAND_CALL(hiprandGenerate(*gen, (uint32_t*)output, count/4)); // (0, 1]
    cnat_compress_cuda_kernel<<<blocks, threads, 0, stream>>>(
            input,
            output,
            count);
}

__global__ void cnat_decompress_cuda_kernel(
    uint8_t* __restrict__ input,
    float* __restrict__ output,
    int len) {
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  if(index < len) {
      uint32_t decode;
      if (input[index]<=127) decode = input[index]+17;
      else decode = input[index]+145;
      if (!input[index] % 128) decode -= 17;
      uint32_t sign_and_exp = decode << 23;
      output[index] = reinterpret_cast<float &>(sign_and_exp);
  }
}

void cnat_decompress(uint8_t* input, float* output, int count, hipStream_t stream) {
  const int threads = 1024;
  auto blocks = count/threads;
  if (count%threads || !blocks) blocks++;
  cnat_decompress_cuda_kernel<<<blocks, threads, 0, stream>>>(
    input,
    output,
    count);
}

