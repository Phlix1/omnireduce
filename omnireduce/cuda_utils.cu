#include "hip/hip_runtime.h"
#include "cuda_utils.hpp"

template <typename scalar_t>
__global__ void bitmap_cuda_kernel(scalar_t* input, uint8_t* bitmap, int64_t len, scalar_t threshold) {
    const auto index = blockIdx.x * blockDim.x + threadIdx.x;
    __shared__ bool zero_block;
    if (threadIdx.x == 0) zero_block = true;
    __syncthreads();
    if(index < len) {
      if(std::abs(input[index]) > threshold) zero_block=false;
    }
    __syncthreads();
    if(index < len) {
      if(zero_block) {
        input[index]=0.0;
        bitmap[blockIdx.x]=1;
      }
      else {
        bitmap[blockIdx.x]=0;
      }
    }
    __syncthreads();
}

void compute_bitmap(float* d_tensor, uint8_t* d_bitmap, int64_t tensor_size, uint32_t block_size, hipStream_t stream, float threshold) {
    uint32_t block_num = tensor_size/block_size;
    if (tensor_size%block_size!=0)
        block_num += 1;
    bitmap_cuda_kernel<<<block_num, block_size, 0, stream>>>(d_tensor, d_bitmap, tensor_size, threshold);
}

void compute_bitmap(int* d_tensor, uint8_t* d_bitmap, int64_t tensor_size, uint32_t block_size, hipStream_t stream, int threshold) {
  uint32_t block_num = tensor_size/block_size;
  if (tensor_size%block_size!=0)
      block_num += 1;
  bitmap_cuda_kernel<<<block_num, block_size, 0, stream>>>(d_tensor, d_bitmap, tensor_size, threshold);
}